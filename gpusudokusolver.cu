
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
int board[16][16] , lock[16][16];
__global__
void possible_move(int sqr , int row,int col ,int move ,int n,int *brd, int *flag){
int tid = threadIdx.x;
if(brd[row*n + tid]==move||brd[tid*n + col]==move||brd[((row/sqr)*sqr+tid/sqr)*n + (col/sqr)*sqr+tid%sqr]==move)
    *flag=1;
}
void print_board(int n){
  int i ,t1;
  for ( i = 0; i < n; i++) {
    for (t1  = 0; t1 < n; t1++) {
      printf("%d  ",board[i][t1]);
    }
    printf("\n");
  }
}
void solve(int n)
{
  int i, j ,k,sqr,*ibd,*flag,size,zero;
  size = sizeof(int);
  sqr=sqrt(n);
  zero=0;
  hipMalloc(&ibd,size*n*n);
  hipMalloc(&flag,size);
  for(i=0;i<n;i++){
    for(j=0;j<n;j++){
      if(board[i][j]==0){
        for(k=0;k<=n;k++){
        hipMemcpy(flag,&zero,size,hipMemcpyHostToDevice);
        for(int t=0; t<n; t++){
            hipMemcpy(ibd+t*n,(int *)board[t],size*n,hipMemcpyHostToDevice);
        }
        possible_move<<<1,n>>>(sqr,i,j,k,n,ibd,flag);
        hipMemcpy(&zero,flag,size,hipMemcpyDeviceToHost);
      if(zero==0)
            {board[i][j]=k;break;}
          else{
            if(k==n){
              if(i==0&&j==0)
                return;
              lock:;
              if(j==0){
                j=n;
                i--;
                if(lock[i][j]==1)
                  goto lock;
                k=board[i][j];
                board[i][j]=0;
              }
              else{
                j--;
                if(lock[i][j]==1)
                  goto lock;
                k=board[i][j];
                board[i][j]=0;
              }
              if(k==n){
                goto lock;
              }
            }
            zero=0;
          }
        }
      }
      printf("\nNextmove %d %d\n",i,j);
      print_board(n);
    }
  }
}
int main(){
  int n,**s,i,p,t1,t2,t3;
  float t4;
  printf("Enter Board Size:");
  while(1){
    scanf("%d",&n);
    t4 = sqrt(n);
    if(n==(int)t4*(int)t4)
      break;
    printf("Enter correct Boards size:");
  }
//Predefined board Numbers
  printf("Enter no. of Predefined numbers:");
  scanf("%d",&p );
  for(i=0;i<p;i++){
    scanf("%d%d%d",&t1,&t2,&t3);
    if(t1<=n&&t2<=n&&t3<=n&&t1>0&&t2>0&&t3>0){
      board[t1-1][t2-1]=t3;
      lock[t1-1][t2-1]=1;
    }
}

  //Print board
  printf("\nInitial Board\n");
  print_board(n);
  //solve board
  solve(n);
  printf("\nFinal Board\n");
  print_board(n);
  return 0;
}
